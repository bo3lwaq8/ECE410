
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <iomanip>

// CUDA kernel to add elements of two arrays
__global__ void add(long long n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

// Helper function to handle CUDA errors
void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(result));
        exit(99);
    }
}

int main(void) {
    // Set up timing events
    hipEvent_t start, stop;
    hipEvent_t kernelStart, kernelStop;
    checkCuda(hipEventCreate(&start));
    checkCuda(hipEventCreate(&stop));
    checkCuda(hipEventCreate(&kernelStart));
    checkCuda(hipEventCreate(&kernelStop));

    std::cout << "Running vector addition for various matrix sizes...\n\n";
    std::cout << std::left << std::setw(12) << "Matrix Size"
              << std::left << std::setw(12) << "N"
              << std::left << std::setw(20) << "Total Time (ms)"
              << std::left << std::setw(20) << "Kernel Time (ms)" << std::endl;
    std::cout << "----------------------------------------------------------------\n";

    // Loop over matrix sizes from 2^15 to 2^25
    for (int i = 15; i <= 25; ++i) {
        long long N = 1LL << i;
        float *x, *y;
        float *d_x, *d_y;
        float total_time_ms, kernel_time_ms;

        // Start timing for total execution
        checkCuda(hipEventRecord(start));

        // Allocate memory on the host (CPU)
        x = new float[N];
        y = new float[N];

        // Allocate memory on the device (GPU)
        checkCuda(hipMalloc(&d_x, N * sizeof(float)));
        checkCuda(hipMalloc(&d_y, N * sizeof(float)));

        // Initialize host arrays
        for (long long j = 0; j < N; j++) {
            x[j] = 1.0f;
            y[j] = 2.0f;
        }

        // Copy data from host to device
        checkCuda(hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));
        checkCuda(hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice));

        // Set up execution configuration
        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;

        // Start timing for kernel execution
        checkCuda(hipEventRecord(kernelStart));

        // Launch the kernel
        add<<<numBlocks, blockSize>>>(N, d_x, d_y);

        // Stop timing for kernel execution
        checkCuda(hipEventRecord(kernelStop));
        
        // Synchronize to make sure kernel is finished before getting results
        checkCuda(hipDeviceSynchronize());

        // Copy results from device to host
        checkCuda(hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost));

        // Stop timing for total execution
        checkCuda(hipEventRecord(stop));
        checkCuda(hipEventSynchronize(stop));

        // Calculate elapsed times
        checkCuda(hipEventElapsedTime(&total_time_ms, start, stop));
        checkCuda(hipEventElapsedTime(&kernel_time_ms, kernelStart, kernelStop));
        
        // Print results for the current size
        std::cout << std::left << std::setw(12) << ("2^" + std::to_string(i))
                  << std::left << std::setw(12) << N
                  << std::left << std::setw(20) << total_time_ms
                  << std::left << std::setw(20) << kernel_time_ms << std::endl;

        // Free device and host memory
        checkCuda(hipFree(d_x));
        checkCuda(hipFree(d_y));
        delete[] x;
        delete[] y;
    }

    // Destroy events
    checkCuda(hipEventDestroy(start));
    checkCuda(hipEventDestroy(stop));
    checkCuda(hipEventDestroy(kernelStart));
    checkCuda(hipEventDestroy(kernelStop));

    return 0;
}
